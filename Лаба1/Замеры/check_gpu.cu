#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>

#define CSC(call)                                                       \
do {                                                                            \
        hipError_t status = call;                              \
        if (status != hipSuccess) {                                                                                                                                                            \
                fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));               \
                exit(0);                                                                \
        }                                                                                       \
} while(0)


__global__ void kernel(double *arr1, double *arr2, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    while(idx < n) {
        arr1[idx] = arr1[idx] - arr2[idx];
        idx += offset;
    }
}

int main() {
    int n;
    FILE *in = fopen("test1_15.txt", "r");
    fscanf(in, "%d", &n);
    double *arr1 = (double *)malloc(sizeof(double) * n);
    for(int i = 0; i < n; i++) {
        fscanf(in,"%lf",&arr1[i]);
    }

    double *arr2 = (double *)malloc(sizeof(double) * n);
    for(int i = 0; i < n; i++) {
        fscanf(in, "%lf",&arr2[i]);
    }
    double *dev_arr1;
    hipMalloc(&dev_arr1, sizeof(double) * n);
    hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice);

    double *dev_arr2;
    hipMalloc(&dev_arr2, sizeof(double) * n);
    hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start));

    kernel<<< 512, 512 >>>(dev_arr1, dev_arr2, n);

    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

    CSC(hipEventRecord(stop));
    CSC(hipEventSynchronize(stop));
    float t;
    CSC(hipEventElapsedTime(&t, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));

    hipMemcpy(arr1, dev_arr1, sizeof(double) * n, hipMemcpyDeviceToHost);
    /*for(int i = 0; i < n; i++){
        printf("%0.10e ", arr1[i]);
    }*/
    printf("%0.10e ", t);

    hipFree(dev_arr1);
    hipFree(dev_arr2);
    free(arr1);
    free(arr2);

    return 0;
}
